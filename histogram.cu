#include "hip/hip_runtime.h"
__global__ void histo_kernal(char *buffer, long size, int *histo )
{
     __shared__ int temp[256];
     temp[threadIdx.x] = 0;
     __syncthreads();

     int i = threadIdx.x + blockIdx.x * blockDim.x;
     int offset = blockDim.x * gridDim.x;
     int z;
     while (i < size)
     {
              z = buffer[i];
              atomicAdd( &temp[z], 1);
              i += offset;
     }
     __syncthreads();


    atomicAdd( &(histo[threadIdx.x]), temp[threadIdx.x] );
}


int main(int argc, char** args)
{

char args_two[10];
char args_four[10];
char *args_first_first;
//char *args_first_second;
//char *args_second_first;
//char *args_second_second;
if((strcmp(args[1], "-g") == 0) && (strcmp(args[3], "-t") == 0))
{
	strcpy(args_two, args[2]);
	args_first_first = strtok(args_two, "x");
	//args_first_second = strtok(NULL, "x");
	strcpy(args_four, args[4]);
	//args_second_first = strtok(args_four, "x");
	//args_second_second = strtok(NULL, "x");
	//printf("%c", array[0]);

}
else
{
printf("Incorrect argument format");
exit(EXIT_FAILURE);
}

// setup cuda timer
hipEvent_t timer_start, timer_stop;
hipEventCreate(&timer_start);
hipEventCreate(&timer_stop);

// start cuda timer
hipEventRecord(timer_start);


FILE *input_file;
char characters[256];
input_file = fopen(args[5], "r");
//unsigned int histo[256];

// set all histogram count values to 0
//int i;
//for(i = 0; i < 256; i++) {
//     histo[i] = 0;
//}


fseek(input_file, 0, SEEK_END); // seek to end of file
long size = ftell(input_file); // get current file pointer
//printf("%d",size);
fseek(input_file, 0, SEEK_SET);
char *host_buffer = (char *) malloc(size + 1);
fgets(host_buffer, size + 1, (FILE*)input_file);
fgets(host_buffer, 1, size + 1, (FILE*)input_file);
//printf(buff);

// create cuda variables and copy host memory to device memory
char *device_buffer;
int *device_histo;
hipMalloc( (void**)&device_buffer, size + 1);
hipMemcpy( device_buffer, host_buffer, size + 1, hipMemcpyHostToDevice );
hipMalloc( (void**)&device_histo,256 * sizeof( long ) );
hipMemset( device_histo, 0, 256 * sizeof( int ) );

// executing kernal
int blocks = atoi(args_first_first);
histo_kernal<<<blocks*8, 1024, 1024*sizeof(int)>>>(device_buffer, size, device_histo);

// copy histogram back to host memory
unsigned int myhisto[256];
hipMemcpy( myhisto, device_histo, 256 * sizeof(int), hipMemcpyDeviceToHost );

// record cuda timing events
hipEventRecord(timer_stop);
hipEventSynchronize(timer_stop);
float ms;
hipEventElapsedTime(&ms, timer_start, timer_stop);
printf("Time Elasped: %3.1f ms\n", ms );

// write histogram to text file
FILE *output_file;
output_file = fopen(args[6], "w");
fprintf(output_file, "Histogram\n");
int w;
for(w = 0; w < 256; w++) {
characters[w] = w;
fprintf(output_file,"%c = %d\n",characters[w], myhisto[w]);
printf("%c = %d\n",characters[w], myhisto[w]);
}

// clean up all cuda events, file pointers, etc
hipEventDestroy(timer_start);
hipEventDestroy(timer_start);
hipFree(device_histo);
hipFree(device_buffer);
free(host_buffer);
fclose(input_file);
fclose(output_file);





}
